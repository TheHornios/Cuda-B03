﻿/**
* ARQUITECTURA DE COMPUTADORES
* 2º Grado en Ingenieria Informatica
*
* Básico 3
*
* Alumno: Rodrigo Pascual Arnaiz
* Fecha: 06/10/2022
*
*/

///////////////////////////////////////////////////////////////////////////
// includes
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h> 

///////////////////////////////////////////////////////////////////////////
// defines
#define BLOQUE_TAM 10

///////////////////////////////////////////////////////////////////////////
// declaracion de funciones
// HOST: funcion llamada desde el host y ejecutada en el host

/**
* Funcion: propiedadesDispositivo
* Objetivo: Mustra las propiedades del dispositvo, esta funcion
* es ejecutada llamada y ejecutada desde el host
*
* Param: INT id_dispositivo -> ID del dispotivo
* Return: cudaDeviceProp -> retorna el onjeto que tiene todas las 
* propiedades del dispositivo CUDA
*/
__host__ hipDeviceProp_t propiedadesDispositivo( int id_dispositivo )
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, id_dispositivo);
	// calculo del numero de cores (SP)
	int cuda_cores = 0;
	int multi_processor_count = deviceProp.multiProcessorCount;
	int major = deviceProp.major;
	int minor = deviceProp.minor;
	switch (major)
	{
	case 1:
		//TESLA
		cuda_cores = 8;
		break;
	case 2:
		//FERMI
		if (minor == 0)
			cuda_cores = 32;
		else
			cuda_cores = 48;
		break;
	case 3:
		//KEPLER
		cuda_cores = 192;
		break;
	case 5:
		//MAXWELL
		cuda_cores = 128;
		break;
	case 6:
		//PASCAL
		cuda_cores = 64;
		break;
	case 7:
		//VOLTA
		cuda_cores = 64;
		break;
	case 8:
		//AMPERE
		cuda_cores = 128;
		break;
	default:
		//DESCONOCIDA
		cuda_cores = 0;
	}
	if (cuda_cores == 0)
	{
		printf("!!!!!dispositivo desconocido!!!!!\n");
	}
	// presentacion de propiedades
	printf("***************************************************\n");
	printf("DISPOSIRIVO %d: %s\n", id_dispositivo, deviceProp.name);
	printf("***************************************************\n");
	printf("> Capacidad de Computo \t\t\t: %d.%d\n", major, minor);
	printf("> N. de MultiProcesadores \t\t: %d \n", multi_processor_count);
	printf("> N. de CUDA Cores (%dx%d) \t\t: %d \n", cuda_cores,
		multi_processor_count, cuda_cores * multi_processor_count);
	printf("> N. max. de Hilos (por bloque) \t: %d \n",
		deviceProp.maxThreadsPerBlock);
	printf(
		" [Eje x\t->\t%d]\n [Eje y\t->\t%d]\n [Eje z\t->\t%d]\n",
		deviceProp.maxThreadsDim[0],
		deviceProp.maxThreadsDim[1],
		deviceProp.maxThreadsDim[2]
	);
	printf("> N. Max. de Bloques (por eje)\n");
	printf(
		" [Eje x\t->\t%d]\n [Eje y\t->\t%d]\n [Eje z\t->\t%d]\n",
		deviceProp.maxGridSize[0],
		deviceProp.maxGridSize[1],
		deviceProp.maxGridSize[2]
	);

	printf("***************************************************\n");
	return deviceProp;
}



/**
* Funcion: rellenarVectorHst
* Objetivo: Funcion que rellena un array pasado por parametro
* con numero aleatorios del 0 al 9
*
* Param: INT* arr -> Puntero del array a rellenar
* Param: INT size -> Longitud del array
* Return: void
*/
__host__ void rellenarVectorHst(int* arr, int size)
{
	for (size_t i = 0; i < size; i++)
	{
		arr[i] = rand() % 10;
	}
}

/**
* Funcion: invertirVector
* Objetivo: Funcion que da la vuelta a un vector pasado por paramtro
*
* Param: INT* arr -> Puntero del array a invertir
* Param: INT size -> Longitud del array
* Return: void
*/
__host__ void invertirVector(int* arr, int size)
{

	int temporal;
	for (int i = 0, x = size - 1; i < x; i++, x--) {
		temporal = arr[i];
		arr[i] = arr[x];
		arr[x] = temporal;
	}

}


/**
* Funcion: sumarArrays
* Objetivo: Funcion que da la vuelta a un vector pasado por paramtro
*
* Param: INT* primer_array -> Primer puntero del array que se quiere sumar
* Param: INT* segundo_array -> Segundo puntero del array que se quiere sumar
* Param: INT* array_resultado -> Puntero del array que va a contener el resultado
* Param: INT size -> Tamaño de los arrays
* Return: void
*/

__global__ void sumarArrays(int* primer_array, int* segundo_array, int* array_resultado, int size)
{
	int idT = threadIdx.x;
	int idB = blockIdx.x;
	int pos = BLOQUE_TAM * idB + idT;
	if (pos < size)
	{
		array_resultado[pos] = primer_array[pos] + segundo_array[pos];
	}
}

///////////////////////////////////////////////////////////////////////////
// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv)
{
	// Semilla de random aleatoria 
	srand( time( NULL ) );

	// Obetener el dispisivo cuda
	int numero_dispositivos;
	hipDeviceProp_t propiedades_dispositivo;
	hipGetDeviceCount(&numero_dispositivos);

	if ( numero_dispositivos == 0 )
	{
		printf("!!!!!ERROR!!!!!\n");
		printf("Este ordenador no tiene dispositivo de ejecucion CUDA\n");
		printf("<pulsa [INTRO] para finalizar>");
		getchar();
		return 1;

	}
	else
	{
		printf("Se han encontrado <%d> dispositivos CUDA:\n", numero_dispositivos);
		for (int id = 0; id < numero_dispositivos; id++)
		{
			propiedades_dispositivo = propiedadesDispositivo(id);
		}
	}

	//************	3. Ejercicio  ************//
	// declaracion de variables
	int* hst_vector1, * hst_vector2, * hst_resultado;
	int* dev_vector1, * dev_vector2, * dev_resultado;


	int numero_bloques; // Número de bloques necesarios
	int numero_elementos;
	bool is_numero_valido = false;
	bool is_cantidad_valida = false;

	do {
		do {
			printf("Introduce el numero de elementos: ");
			is_numero_valido = scanf("%i", &numero_elementos);
			printf("\n");
		} while (!is_numero_valido);


		numero_bloques = ceil( (float)numero_elementos / (float)BLOQUE_TAM );
		printf("Utilizando %i bloques de %i hilos (%i hilos)\n\n", numero_bloques, BLOQUE_TAM, numero_bloques* BLOQUE_TAM);
		if ( numero_bloques < propiedades_dispositivo.maxGridSize[0]  )
		{
			is_cantidad_valida = true;
		}
		else {
			printf("> ERROR: numero maximo de bloques superado! [ %d bloques]\n", propiedades_dispositivo.maxGridSize[0]);
		}

	} while (!is_cantidad_valida);

	printf("> Vector de %d elementos \n", numero_elementos);

	
	// reserva de memoria en el host
	hst_vector1 = (int*)malloc(numero_elementos * sizeof(int));
	hst_vector2 = (int*)malloc(numero_elementos * sizeof(int));
	hst_resultado = (int*)malloc(numero_elementos * sizeof(int));

	// reserva de memoria en el device
	hipMalloc((void**)&dev_vector1, numero_elementos * sizeof(int));
	hipMalloc((void**)&dev_vector2, numero_elementos * sizeof(int));
	hipMalloc((void**)&dev_resultado, numero_elementos * sizeof(int));



	// Rellenamos el vector con la funcion previamente creada 
	rellenarVectorHst( hst_vector1, numero_elementos);

	
	// Copiamos el vector hst 1 al vector hst 2, la ide es invertir la copia en hst 2
	hipMemcpy( hst_vector2, hst_vector1, numero_elementos * sizeof(int), hipMemcpyHostToHost );

	// Invertimos el vector y ese mismo vector es el resultado
	invertirVector(hst_vector2, numero_elementos );

	// Copiamos el vextor invertido en  h
	hipMemcpy( dev_vector2, hst_vector2, numero_elementos * sizeof(int), hipMemcpyHostToDevice );

	// Copiamos el contenido del vector device 2 al vector host 2
	hipMemcpy( hst_vector2, dev_vector2, numero_elementos * sizeof(int), hipMemcpyDeviceToHost);


	// Mostrar vector 1
	printf("VECTOR 1:\n");
	for (int i = 0; i < numero_elementos; i++)
	{
		printf("%i ", hst_vector1[i]);
	}
	printf("\n");
	// Mostrar vector 2
	printf("VECTOR 2:\n");
	for (int i = 0; i < numero_elementos; i++)
	{
		printf("%i ", hst_vector2[i]);
	}
	printf("\n");


	// La suma será realizada en el device aprovechando todos los hilos o threads lanzados sin sobrepasar el máximo permitido.


	// Sumar V1 + V2
	hipMemcpy(dev_vector1, hst_vector1, numero_elementos * sizeof(int), hipMemcpyHostToDevice);
	sumarArrays <<<numero_bloques, BLOQUE_TAM >>> (dev_vector1, dev_vector2, dev_resultado, numero_elementos);
	hipMemcpy(hst_resultado, dev_resultado, numero_elementos * sizeof(int), hipMemcpyDeviceToHost);

	// Mostrar resultado de la suma
	printf("\nSUMA:\n", numero_elementos);
	for (int i = 0; i < numero_elementos; i++)
	{
		printf("%i ", hst_resultado[i]);
	}
	printf("\n");

	// Salida del programa
	time_t fecha;
	time(&fecha);
	printf("***************************************************\n");
	printf("Programa ejecutado el: %s\n", ctime(&fecha));
	printf("<pulsa [INTRO] para finalizar>");
	getchar();
	return 0;

}
///////////////////////////////////////////////////////////////////////////